
#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdlib.h>


#define THREAD_PER_BLOCK 256

//1. cuda.h 和cuda_runtime ?

__global__ void reduce1(float* d_intput,float * d_output){
    __shared__ float input_begin[THREAD_PER_BLOCK];

    float *input_begin_global=d_intput+blockDim.x * blockIdx.x;
    input_begin[threadIdx.x]=input_begin_global[threadIdx.x];
    __syncthreads();

    for(int idx=1;idx<blockDim.x;idx=idx*2){
        if(threadIdx.x%(idx*2) == 0)
            input_begin[threadIdx.x]+= input_begin[threadIdx.x+idx];
        __syncthreads();
    }
    
    if(threadIdx.x==0)
        d_output[blockIdx.x]=input_begin[0];

}
bool check(float* a,float*b,int n){
    for(int i=0;i<n;i++){
        if(abs(a[i]-b[i])>0.005) return false;
    }
    return true;
}
void printArray(float* a,int n){
    for(int i=0;i<10;i++){
        printf("%f ",a[i]);
    }
    printf("\n");
}



int main(){
    const int N=32 * 1024 * 1024;
    float* h_input= (float*)malloc(N*sizeof(float));
    float* d_intput;
    hipMalloc((void**)&d_intput,N*sizeof(float));
    
    constexpr int block_num=(N+THREAD_PER_BLOCK-1)/THREAD_PER_BLOCK;
    float *h_output= (float*)malloc(block_num*sizeof(float));
   
    float *d_output;
    hipMalloc((void**)&d_output,block_num*sizeof(float));

    float* result=(float*) malloc(block_num*sizeof(float));
    

    {
        for(int i=0;i<N;i++){
            h_input[i]=2.0*drand48()-1.0;
        }
    }

    // compute on cpu side
    {
        for(int i=0;i<block_num;i++){
            float tsum=0;
            for(int j=0;j<THREAD_PER_BLOCK;j++){
                tsum+=h_input[THREAD_PER_BLOCK*i+j];
            }
            h_output[i]=tsum;
        }
    }
    // compute on GPU
    {
        hipMemcpy(d_intput,h_input,N*sizeof(float),hipMemcpyHostToDevice);
        dim3 Grid(block_num,1);
        dim3 Block(THREAD_PER_BLOCK,1);
        
        reduce1<<<Grid,Block>>>(d_intput,d_output);
    }
    //check & release
    {
        
        hipMemcpy(result,d_output,block_num*sizeof(float),hipMemcpyDeviceToHost);

        if(check(result,h_output,block_num)){
            printf("ans is good\n");
        }else{
            printf("ans is wrong\n");
            printArray(result,block_num);
            printArray(h_output,block_num);
            
        }

        hipFree(d_intput);
        hipFree(d_output);
        
        free(h_input);
        free(result);
        free(h_output);
    }

    return 0;
    
}