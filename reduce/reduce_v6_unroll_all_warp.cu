
#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdlib.h>


#define THREAD_PER_BLOCK 256
#define WARP_SIZE 32
/**
 * 
 */
__global__ void reduce2(float* d_intput,float * d_output){
    float *input_begin_global=d_intput+blockDim.x * blockIdx.x *2;
    __shared__ float input_begin[THREAD_PER_BLOCK];
    input_begin[threadIdx.x]=input_begin_global[threadIdx.x]+ input_begin_global[threadIdx.x+THREAD_PER_BLOCK];
    
    __syncthreads();

    //完全展开for循环，可以通过编译器的宏
    {
        #pragma unroll
        for(int interval=blockDim.x/2;interval>0;interval/=2){
            if(threadIdx.x<interval){
                input_begin[threadIdx.x]+=input_begin[threadIdx.x+interval];
            }
            if(interval>WARP_SIZE)
                __syncthreads();
        }

    }
    
    if(threadIdx.x==0)
        d_output[blockIdx.x]=input_begin[0];

}
bool check(float* a,float*b,int n){
    for(int i=0;i<n;i++){
        if(abs(a[i]-b[i])>0.005) return false;
    }
    return true;
}
void printArray(float* a,int n){
    for(int i=0;i<10;i++){        
        printf("%f ",a[i]);
    }
    printf("\n");
}



int main(){
    const int N=32 * 1024 * 1024;
    float* h_input= (float*)malloc(N*sizeof(float));
    float* d_intput;
    hipMalloc((void**)&d_intput,N*sizeof(float));
    
    constexpr int block_num=(N+THREAD_PER_BLOCK-1)/THREAD_PER_BLOCK/2;
    float *h_output= (float*)malloc(block_num*sizeof(float));
   
    float *d_output;
    hipMalloc((void**)&d_output,block_num*sizeof(float));

    float* result=(float*) malloc(block_num*sizeof(float));
    

    {
        for(int i=0;i<N;i++){
            h_input[i]=2.0*drand48()-1.0;
        }
    }

    // compute on cpu side
    {
        for(int i=0;i<block_num;i++){
            float tsum=0;
            for(int j=0;j<2 * THREAD_PER_BLOCK;j++){
                tsum+=h_input[ 2 * THREAD_PER_BLOCK*i+j];
            }
            h_output[i]=tsum;
        }
    }
    // compute on GPU
    {
        hipMemcpy(d_intput,h_input,N*sizeof(float),hipMemcpyHostToDevice);
        dim3 Grid(block_num,1);
        dim3 Block(THREAD_PER_BLOCK,1);
        
        reduce2<<<Grid,Block>>>(d_intput,d_output);
    }
    //check & release
    {
        
        hipMemcpy(result,d_output,block_num*sizeof(float),hipMemcpyDeviceToHost);

        if(check(result,h_output,block_num)){
            printf("ans is good\n");
        }else{
            printf("ans is wrong\n");
            printArray(result,block_num);
            printArray(h_output,block_num);
            
        }

        hipFree(d_intput);
        hipFree(d_output);
        
        free(h_input);
        free(result);
        free(h_output);
    }

    return 0;
    
}