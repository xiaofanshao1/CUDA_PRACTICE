#include "hip/hip_runtime.h"

#include <iostream>
#include <hip/hip_runtime.h>
// M=2048 N=512
// thread&data block[32,8] ->  grid[64,64]
__global__ void transpose_naive_v0(float* input, float* output)
{

    //transpose符合矩阵分块规律 
    
    //1. 思路一：整体一起考虑，成为一个大的block（因为没有引入sharem mem，可以跨越原来分的block划分，作为一个大整体来进行操作
    //2. 思路二：分块+块内操作
    //for block wise:  swap row&col
    //for block internal: swap row&col

    //下面采用大整体操作，合并所有block，在大整体上进行操作
    int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (col_idx < MATRIX_N && row_idx < MATRIX_M) {

       int idx = row_idx * MATRIX_N + col_idx;
       int trans_idx = col_idx * MATRIX_M + row_idx;
       output[trans_idx] = input[idx];
    }
}
void transpose_cpu(float *input,float *output, const int M,const int N){
    for(int m=0;m<M;m++){ 
        for(int n=0;n<N;n++){
            const int input_index=m*N+n;
            const int output_index=n*M+m;
            output[output_index]=intput[input_index];
        }
    }
}

class Perf{
private:
    std::string m_name;
    hipEvent_t m_start,m_end;
public:
    Perf(const std::string *name){
        m_name=name;
        hipEventCreate(&m_start);
        hipEventCreate(&m_end);
        hipEventRecord(m_start);
        hipEventSynchronize(m_start);

    }
    ~Perf(){
        hipEventRecord(m_end);
        hipEventSynchronize(m_end);
        float elapse_time=0.0;
        hipEventElapsedTime(&elapsed_time,m_start,m_end);
        std::cout<<m_name<<"elapse:"<<elapsed_time<<" ms"<<std::endl;
    }
}

int main(){
    const int MATRIX_M=2048;
    const int MATRIX_N=512;
    const size_t size= MATRIX_M *MATRIX_N;

    float *input_host=(float *)malloc(size*sizeof(float));
    float *output_host_cpu_calc=(float*)malloc(size*sizeof(float));
    float *output_host_gpu_calc=(float*)malloc(size*sizeof(float));

    for(int i=0;i<size;i++){
        input_host[i]= 2.0 *(float)drand8()-1.0;
    }

    transpose_cpu(input_host,output_host_cpu_calc,MATRIX_M,MATRIX_N);
    float *input_device,*output_device;

    hipMalloc();
    hipMemcpy();
    hipMalloc();

    //=============
    hipMemset(output_device,0,size*sizeof(float));

    for(int i=0;i<5;i++){
        Perf perf("transepose_native_32_8");
        dim3 block_size(32,8);
        dim3 grid_size();
        transpose_naive_v0<<<grid_size,block_size>>>(input_device,output-device,MATRIX_M,MATRIX_N);
        cudaDeviceSyncchronize();
    }
    hipMemcpy(output_host_gpu_calc,output_device,
                size*sizeof(float),hipMemcpyDeviceToHost);
    if(){
        std::cout<<"right"<<std::end;
    }else{
        std::cout<<"wrong"<<std::end; 
    }


}