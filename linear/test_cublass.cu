#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        std::cerr << "CUDA Error at " << __FILE__ << ":" << __LINE__ << ": "   \
                  << hipGetErrorString(status) << std::endl;                  \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
}

#define CHECK_CUBLAS(func)                                                     \
{                                                                              \
    hipblasStatus_t status = (func);                                            \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
        std::cerr << "CUBLAS Error at " << __FILE__ << ":" << __LINE__ << ": " \
                  << status << std::endl;                                      \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
}

int main() {
    // 矩阵维度 (M=512, N=512, K=1024)
    const int M = 512, N = 512, K = 1024;

    // 分配主机内存 (A[M,K], B[K,N], C[M,N])
    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_C = new float[M * N];

    // 初始化数据 (示例: A全1, B全2)
    std::fill(h_A, h_A + M * K, 1.0f);
    std::fill(h_B, h_B + K * N, 2.0f);

    // 分配设备内存
    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, M * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B, K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, M * N * sizeof(float)));

    // 拷贝数据到设备
    CHECK_CUDA(hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));

    // 创建 cuBLAS 句柄
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // 执行 GEMM: C = alpha * A * B + beta * C
    const float alpha = 1.0f, beta = 0.0f;
    CHECK_CUBLAS(hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,  // 不转置A和B
        M, N, K,                    // 矩阵维度
        &alpha,                     // alpha
        d_A, HIP_R_32F, M,         // A矩阵 (行主序)
        d_B, HIP_R_32F, K,         // B矩阵 (行主序)
        &beta,                      // beta
        d_C, HIP_R_32F, M,         // C矩阵 (行主序)
        HIP_R_32F,                 // 计算精度
        HIPBLAS_GEMM_DEFAULT         // 算法选择
    ));

    // 拷贝结果回主机
    CHECK_CUDA(hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // 验证结果 (C[0,0] 应为 1*2*1024=2048)
    std::cout << "C[0] = " << h_C[0] << " (预期: 2048)" << std::endl;

    // 释放资源
    delete[] h_A; delete[] h_B; delete[] h_C;
    CHECK_CUDA(hipFree(d_A)); CHECK_CUDA(hipFree(d_B)); CHECK_CUDA(hipFree(d_C));
    CHECK_CUBLAS(hipblasDestroy(handle));

    return 0;
}
