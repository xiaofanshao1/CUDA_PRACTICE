#include "hip/hip_runtime.h"
// Adapted from https://github.com/luliyucoordinate/cute-flash-attention/blob/main/flash.cu ffb53fd495a8ac5d079a1f3382622d018a0d50f7
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/types.h>

#include <cute/tensor.hpp>

template <typename config>
__global__ void flash_forward(void* output, const void* q, const void* k,
                              const void* v, int head_stride, int q_len,
                              int k_len, float sm_scale) {
  using namespace cute;
  using X = Underscore;
  const int m_block = blockIdx.x;
  const int base_id = blockIdx.y;
  const int tidx = threadIdx.x;

  using T = typename config::T;
  using SmemLayoutQ = typename config::SmemLayoutQ;
  using SmemLayoutK = typename config::SmemLayoutKV;
  using SmemLayoutV = typename config::SmemLayoutKV;
  using SmemLayoutO = typename config::SmemLayoutO;
  using SmemCopyAtom = typename config::SmemCopyAtom;
  using SmemCopyAtomO = typename config::SmemCopyAtomO;
  using GmemTiledCopyQKV = typename config::GmemTiledCopyQKV;
  using GmemTiledCopyO = typename config::GmemTiledCopyO;
  using SmemCopyAtomTransposed = typename config::SmemCopyAtomTransposed;
  using TiledMMA = typename config::TiledMMA;
  using SmemLayoutVt = typename config::SmemLayoutVtransposed;
  using SmemLayoutVtNoSwizzle = typename config::SmemLayoutVtransposedNoSwizzle;

  constexpr int kBlockM = config::kBlockM;
  constexpr int kBlockN = config::kBlockN;
  constexpr int kHeadDim = config::kHeadDim;

  extern __shared__ T shm_data[];
  auto q_shm = shm_data;
  auto k_shm = q_shm + cosize(SmemLayoutQ{});
  auto v_shm = k_shm + cosize(SmemLayoutK{});

  const int bs_head_offset = base_id * head_stride;

  auto Q = make_tensor(make_gmem_ptr<half_t>((T*)q + bs_head_offset),
                       make_shape(q_len, Int<kHeadDim>{}),
                       make_stride(Int<kHeadDim>{}, Int<1>{}));
  auto K = make_tensor(make_gmem_ptr<half_t>((T*)k + bs_head_offset),
                       make_shape(k_len, Int<kHeadDim>{}),
                       make_stride(Int<kHeadDim>{}, Int<1>{}));
  auto V = make_tensor(make_gmem_ptr<half_t>((T*)v + bs_head_offset),
                       make_shape(k_len, Int<kHeadDim>{}),
                       make_stride(Int<kHeadDim>{}, Int<1>{}));
  auto O = make_tensor(make_gmem_ptr<half_t>((T*)output + bs_head_offset),
                       make_shape(q_len, Int<kHeadDim>{}),
                       make_stride(Int<kHeadDim>{}, Int<1>{}));

  auto gQ = local_tile(Q, make_tile(Int<kBlockM>{}, Int<kHeadDim>{}),
                       make_coord(m_block, _));
  auto gK = local_tile(K, make_tile(Int<kBlockN>{}, Int<kHeadDim>{}),
                       make_coord(0, _));
  auto gV = local_tile(V, make_tile(Int<kBlockN>{}, Int<kHeadDim>{}),
                       make_coord(0, _));

  auto sQ = make_tensor(make_smem_ptr<half_t>(q_shm), SmemLayoutQ{});
  auto sK = make_tensor(make_smem_ptr<half_t>(k_shm), SmemLayoutK{});
  auto sV = make_tensor(make_smem_ptr<half_t>(v_shm), SmemLayoutV{});

  // Tensor for V Transpose; used in GEMM-II.
  auto sVt = make_tensor(make_smem_ptr<half_t>(v_shm), SmemLayoutVt{});
  auto sVtNoSwizzle =
      make_tensor(make_smem_ptr<half_t>(v_shm), SmemLayoutVtNoSwizzle{});

  GmemTiledCopyQKV gmem_tiled_copy_QKV;
  auto gmem_thr_copy_QKV = gmem_tiled_copy_QKV.get_thread_slice(tidx);
  auto tQgQ = gmem_thr_copy_QKV.partition_S(gQ(_, _, 0));
  auto tQsQ = gmem_thr_copy_QKV.partition_D(sQ);
  auto tKgK = gmem_thr_copy_QKV.partition_S(gK(_, _, 0));
  auto tKsK = gmem_thr_copy_QKV.partition_D(sK);
  auto tVgV = gmem_thr_copy_QKV.partition_S(gV(_, _, 0));
  auto tVsV = gmem_thr_copy_QKV.partition_D(sV);

  TiledMMA tiled_mma;
  auto thr_mma = tiled_mma.get_slice(tidx);
  auto tSrQ = thr_mma.partition_fragment_A(sQ);             // (MMA,MMA_M,MMA_K)
  auto tSrK = thr_mma.partition_fragment_B(sK);             // (MMA,MMA_N,MMA_K)
  auto tOrVt = thr_mma.partition_fragment_B(sVtNoSwizzle);  // (MMA,MMA_K,MMA_N)

  auto smem_tiled_copy_Q = make_tiled_copy_A(SmemCopyAtom{}, tiled_mma);
  auto smem_thr_copy_Q = smem_tiled_copy_Q.get_thread_slice(tidx);
  auto tSsQ = smem_thr_copy_Q.partition_S(sQ);
  auto tSrQ_view = smem_thr_copy_Q.retile_D(tSrQ);

  auto smem_tiled_copy_K = make_tiled_copy_B(SmemCopyAtom{}, tiled_mma);
  auto smem_thr_copy_K = smem_tiled_copy_K.get_thread_slice(tidx);
  auto tSsK = smem_thr_copy_K.partition_S(sK);
  auto tSrK_view = smem_thr_copy_K.retile_D(tSrK);

  auto smem_tiled_copy_V =
      make_tiled_copy_B(SmemCopyAtomTransposed{}, tiled_mma);
  auto smem_thr_copy_V = smem_tiled_copy_V.get_thread_slice(tidx);
  auto tOsVt = smem_thr_copy_V.partition_S(sVt);
  auto tOrVt_view = smem_thr_copy_V.retile_D(tOrVt);

  // copy q
  cute::copy(gmem_tiled_copy_QKV, tQgQ, tQsQ);
  cp_async_fence();
  cp_async_wait<0>();
  __syncthreads();

  // multiply sm scale
  half2 sm_half2 = {__float2half_rn(sm_scale), __float2half_rn(sm_scale)};
  auto tQsQ_int4 = recast<int4>(tQsQ);
#pragma unroll
  for (int ii = 0; ii < size(tQsQ_int4); ii++) {
    auto tmp = tQsQ_int4(ii);
    auto tmp_half2 = (half2*)&tmp;
#pragma unroll
    for (int jj = 0; jj < 4; jj++) {
      tmp_half2[jj] = __hmul2_rn(sm_half2, tmp_half2[jj]);
    }
    tQsQ_int4(ii) = tmp;
  }
  // multiply sm scale

  // copy kv
  cute::copy(gmem_tiled_copy_QKV, tKgK, tKsK);
  cp_async_fence();
  cute::copy(gmem_tiled_copy_QKV, tVgV, tVsV);
  cp_async_fence();
  // copy kv

  // ((2,2),MMA_M,MMA_K)
  auto rAccOut =
      partition_fragment_C(tiled_mma, Shape<Int<kBlockM>, Int<kHeadDim>>{});
  auto scores_max =
      make_tensor<float>(Shape<Int<2 * size<1>(rAccOut)>>{});  // (2*MMA_M)
  auto scores_sum = make_fragment_like(scores_max);
  auto rAccScore = partition_fragment_C(
      tiled_mma, make_shape(Int<kBlockM>{}, Int<kBlockN>{}));
  clear(rAccOut);
  // init scores_max, scores_sum
#pragma unroll
  for (int ii = 0; ii < size(scores_max); ii++) {
    scores_max(ii) = float(-5e4);
    scores_sum(ii) = 0;
  }

  // ((2,2),MMA_M,MMA_N) to ((2,MMA_M),(2,MMA_N))
  auto ol = logical_divide(rAccOut.layout(), Shape<Int<2>>{});
  auto rAccOut_new_layout =
      make_layout(make_layout(get<1>(get<0>(ol)), get<1>(ol)),
                  make_layout(get<0>(get<0>(ol)), get<2>(ol)));
  auto rAccOut_new = make_tensor(rAccOut.data(), rAccOut_new_layout);

  const int n_block_min = 0;
  int n_block_max = cute::ceil_div(k_len, kBlockN);
#pragma unroll 1
  for (int ii = n_block_min; ii < n_block_max; ii++) {
    clear(rAccScore);
    // wait k
    cp_async_wait<1>();
    __syncthreads();

    // S = Q@K.T
    cute::copy(smem_tiled_copy_Q, tSsQ(_, _, Int<0>{}),
               tSrQ_view(_, _, Int<0>{}));
    cute::copy(smem_tiled_copy_K, tSsK(_, _, Int<0>{}),
               tSrK_view(_, _, Int<0>{}));
#pragma unroll
    for (int si = 0; si < size<2>(tSrQ); si++) {
      if (si < size<2>(tSrQ) - 1) {
        cute::copy(smem_tiled_copy_Q, tSsQ(_, _, si + 1),
                   tSrQ_view(_, _, si + 1));
        cute::copy(smem_tiled_copy_K, tSsK(_, _, si + 1),
                   tSrK_view(_, _, si + 1));
      }
      cute::gemm(tiled_mma, tSrQ(_, _, si), tSrK(_, _, si), rAccScore);
    }

    // ((2, 2),(MMA_M, MMA_N)) -> ((2,MMA_M),(2,MMA_N))
    auto sl = logical_divide(rAccScore.layout(), Shape<Int<2>>{});
    auto rAccScore_new_layout =
        make_layout(make_layout(get<1>(get<0>(sl)), get<1>(sl)),
                    make_layout(get<0>(get<0>(sl)), get<2>(sl)));
    auto scores = make_tensor(rAccScore.data(), rAccScore_new_layout);

    // softmax
    auto scores_max_pre = make_fragment_like(scores_max);
    cute::copy(scores_max, scores_max_pre);
#pragma unroll
    for (int si = 0; si < size<0>(scores); si++) {
      float& scores_max_si = scores_max(si);
      float& scores_sum_si = scores_sum(si);
#pragma unroll
      for (int sj = 0; sj < size<1>(scores); sj++) {
        scores_max_si = max(scores_max_si, scores(si, sj));
      }
      scores_max_si =
          max(scores_max_si, __shfl_xor_sync(0xffffffff, scores_max_si, 0x2));
      scores_max_si =
          max(scores_max_si, __shfl_xor_sync(0xffffffff, scores_max_si, 0x1));

      float scores_scale = exp2f(scores_max_pre(si) - scores_max_si);
#pragma unroll
      for (int sj = 0; sj < size<1>(rAccOut_new); sj++) {
        rAccOut_new(si, sj) *= scores_scale;
      }

      float scores_sum_cur_si = 0;
#pragma unroll
      for (int sj = 0; sj < size<1>(scores); sj++) {
        scores(si, sj) = exp2f(scores(si, sj) - scores_max_si);
        scores_sum_cur_si += scores(si, sj);
      }
      scores_sum_cur_si += __shfl_xor_sync(0xffffffff, scores_sum_cur_si, 0x2);
      scores_sum_cur_si += __shfl_xor_sync(0xffffffff, scores_sum_cur_si, 0x1);
      scores_sum_si = scores_sum_si * scores_scale + scores_sum_cur_si;
    }

    __syncthreads();
    // advance k
    if (ii != n_block_max - 1) {
      gK = local_tile(K, make_tile(Int<kBlockN>{}, Int<kHeadDim>{}),
                      make_coord(ii + 1, _));
      tKgK = gmem_thr_copy_QKV.partition_S(gK(_, _, 0));
      cute::copy(gmem_tiled_copy_QKV, tKgK, tKsK);
    }
    cp_async_fence();
    // wait v
    cp_async_wait<1>();
    __syncthreads();

    // O = softmax(S)*V
    auto scores_fp16 = make_tensor_like<half_t>(scores);
    auto scores_fp32x2 = recast<float2>(scores);
    auto scores_fp16x2 = recast<half2>(scores_fp16);
#pragma unroll
    for (int si = 0; si < size(scores_fp16x2); si++) {
      scores_fp16x2(si) = __float22half2_rn(scores_fp32x2(si));
    }
    // ((2,MMA_M),(2,MMA_N)) to ((2,2,2),MMA_M,MMA_N/2)
    // ((2,MMA_M),(2,(2,MMA_N/2)))
    auto l = logical_divide(scores.layout(), Shape<X, Shape<X, Int<2>>>{});
    auto scores_new_layout =
        make_layout(make_layout(get<0>(get<1>(l)), get<0>(get<0>(l)),
                                get<0>(get<1>(get<1>(l)))),
                    get<1>(get<0>(l)), get<1>(get<1>(get<1>(l))));
    auto tOrS = make_tensor(scores_fp16.data(), scores_new_layout);

    cute::copy(smem_tiled_copy_V, tOsVt(_, _, Int<0>{}),
               tOrVt_view(_, _, Int<0>{}));
#pragma unroll
    for (int oi = 0; oi < size<2>(tOrS); oi++) {
      if (oi < size<2>(tOrS) - 1) {
        cute::copy(smem_tiled_copy_V, tOsVt(_, _, oi + 1),
                   tOrVt_view(_, _, oi + 1));
      }
      cute::gemm(tiled_mma, tOrS(_, _, oi), tOrVt(_, _, oi), rAccOut);
    }

    __syncthreads();
    if (ii != n_block_max - 1) {
      gV = local_tile(V, make_tile(Int<kBlockN>{}, Int<kHeadDim>{}),
                      make_coord(ii + 1, _));
      tVgV = gmem_thr_copy_QKV.partition_S(gV(_, _, 0));
      cute::copy(gmem_tiled_copy_QKV, tVgV, tVsV);
    }
    cp_async_fence();
  }

  // normalize d
#pragma unroll
  for (int si = 0; si < size(scores_sum); si++) {
    scores_sum(si) = __frcp_rn(scores_sum(si));
  }
#pragma unroll
  for (int oi = 0; oi < size<0>(rAccOut_new); oi++) {
#pragma unroll
    for (int oj = 0; oj < size<1>(rAccOut_new); oj++) {
      rAccOut_new(oi, oj) *= scores_sum(oi);
    }
  }

  // write back
  auto rAccOut_fp16 = make_tensor_like<half_t>(rAccOut);
  auto rAccOut_fp32x2 = recast<float2>(rAccOut);
  auto rAccOut_fp16x2 = recast<half2>(rAccOut_fp16);
#pragma unroll
  for (int si = 0; si < size(rAccOut_fp16x2); si++) {
    rAccOut_fp16x2(si) = __float22half2_rn(rAccOut_fp32x2(si));
  }

  auto sO = make_tensor(sQ.data(), SmemLayoutO{});
  auto smem_tiled_copy_O = make_tiled_copy_C(SmemCopyAtomO{}, tiled_mma);
  auto smem_thr_copy_O = smem_tiled_copy_O.get_thread_slice(tidx);
  // ((Atom,AtomNum),MMA_M,MMA_N)
  auto taccOrO = smem_thr_copy_O.retile_S(rAccOut_fp16);
  // ((Atom,AtomNum),PIPE_M,PIPE_N)
  auto taccOsO = smem_thr_copy_O.partition_D(sO);
  cute::copy(smem_tiled_copy_O, taccOrO, taccOsO);

  auto gO = local_tile(O, make_tile(Int<kBlockM>{}, Int<kHeadDim>{}),
                       make_coord(m_block, _));
  GmemTiledCopyO gmem_tiled_copy_O;
  auto gmem_thr_copy_O = gmem_tiled_copy_O.get_thread_slice(tidx);
  // ((Atom,AtomNum),ATOM_M,ATOM_N)
  auto tOsO = gmem_thr_copy_O.partition_S(sO);
  auto tOgO = gmem_thr_copy_O.partition_D(gO(_, _, 0));

  __syncthreads();
  cute::copy(gmem_tiled_copy_O, tOsO, tOgO);
}

namespace config {
using namespace cute;

template <typename T_, int kHeadDim_ = 64, int kBlockM_ = 64, int kBlockN_ = 64>
struct FlashConfig {
  using T = T_;
  static constexpr int kHeadDim = kHeadDim_;
  static constexpr int kBlockM = kBlockM_;
  static constexpr int kBlockN = kBlockN_;

  static constexpr int kBlockKSmem = kHeadDim % 64 == 0 ? 64 : 32;
  static constexpr int kBlockKGmem =
      kHeadDim % 128 == 0 ? 128 : (kHeadDim % 64 == 0 ? 64 : 32);
  static constexpr int kSwizzle = kBlockKSmem == 32 ? 2 : 3;
  using SmemLayoutAtom = decltype(composition(
      Swizzle<kSwizzle, 3, 3>{}, Layout<Shape<Int<8>, Int<kBlockKSmem>>,
                                        Stride<Int<kBlockKSmem>, Int<1>>>{}));
  using SmemLayoutQ = decltype(tile_to_shape(
      SmemLayoutAtom{}, Shape<Int<kBlockM>, Int<kHeadDim>>{}));
  using SmemLayoutKV = decltype(tile_to_shape(
      SmemLayoutAtom{}, Shape<Int<kBlockN>, Int<kHeadDim>>{}));

  using SmemLayoutAtomVtransposedNoSwizzle =
      Layout<Shape<Int<kBlockKSmem>, Int<kBlockN>>,
             Stride<Int<1>, Int<kBlockKSmem>>>;
  using SmemLayoutAtomVtransposed = decltype(composition(
      Swizzle<kSwizzle, 3, 3>{}, SmemLayoutAtomVtransposedNoSwizzle{}));
  using SmemLayoutVtransposed = decltype(tile_to_shape(
      SmemLayoutAtomVtransposed{}, Shape<Int<kHeadDim>, Int<kBlockN>>{}));
  using SmemLayoutVtransposedNoSwizzle =
      decltype(tile_to_shape(SmemLayoutAtomVtransposedNoSwizzle{},
                             Shape<Int<kHeadDim>, Int<kBlockN>>{}));

  using SmemCopyAtom = Copy_Atom<SM75_U32x4_LDSM_N, T>;
  using SmemCopyAtomTransposed = Copy_Atom<SM75_U16x8_LDSM_T, T>;
  using SmemLayoutAtomO = decltype(composition(
      Swizzle<kSwizzle, 3, 3>{}, Layout<Shape<Int<8>, Int<kBlockKSmem>>,
                                        Stride<Int<kBlockKSmem>, Int<1>>>{}));
  using SmemLayoutO = decltype(tile_to_shape(
      SmemLayoutAtomO{}, Shape<Int<kBlockM>, Int<kHeadDim>>{}));
  using SmemCopyAtomO = Copy_Atom<DefaultCopy, T>;

  using mma_op = SM80_16x8x16_F32F16F16F32_TN;
  using mma_traits = MMA_Traits<mma_op>;
  using mma_atom = MMA_Atom<mma_traits>;
  static constexpr int kMmaEURepeatM = 4;
  static constexpr int kMmaEURepeatN = 1;
  static constexpr int kMmaEURepeatK = 1;

  using mma_atom_shape = mma_traits::Shape_MNK;
  static constexpr int kMmaPM = 1 * kMmaEURepeatM * get<0>(mma_atom_shape{});
  static constexpr int kMmaPN = 2 * kMmaEURepeatN * get<1>(mma_atom_shape{});
  static constexpr int kMmaPK = 1 * kMmaEURepeatK * get<2>(mma_atom_shape{});

  using MMA_EU_RepeatT = decltype(make_layout(make_shape(
      Int<kMmaEURepeatM>{}, Int<kMmaEURepeatN>{}, Int<kMmaEURepeatK>{})));
  using MMA_P_T = Tile<Int<kMmaPM>, Int<kMmaPN>, Int<kMmaPK>>;

  using TiledMMA =
      decltype(make_tiled_mma(mma_atom{}, MMA_EU_RepeatT{}, MMA_P_T{}));
  static constexpr int kThreadNum = size(TiledMMA{});
  using g2s_copy_op = SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;
  using g2s_copy_traits = Copy_Traits<g2s_copy_op>;
  static constexpr int kGmemThreadsPerRow = kBlockKSmem / 8;
  using gmem_copy_atom = Copy_Atom<g2s_copy_traits, cute::half_t>;
  using gmem_thr_layout = Layout<
      Shape<Int<kThreadNum / kGmemThreadsPerRow>, Int<kGmemThreadsPerRow>>,
      Stride<Int<kGmemThreadsPerRow>, Int<1>>>;
  using gmem_val_layout = Layout<Shape<Int<1>, Int<8>>>;
  using GmemTiledCopyQKV = decltype(make_tiled_copy(
      gmem_copy_atom{}, gmem_thr_layout{}, gmem_val_layout{}));
  using s2g_copy_atom = Copy_Atom<UniversalCopy<cute::uint128_t>, T>;
  using GmemTiledCopyO = decltype(make_tiled_copy(
      s2g_copy_atom{}, gmem_thr_layout{}, gmem_val_layout{}));

  static constexpr int shm_size_q = cute::cosize(SmemLayoutQ{});
  static constexpr int shm_size_kv = cute::cosize(SmemLayoutKV{}) * 2;
  static constexpr int kShmSize = (shm_size_kv + shm_size_q) * sizeof(half);
};

}  // namespace config

torch::Tensor forward(torch::Tensor q, torch::Tensor k, torch::Tensor v) {
  int bs = q.size(0);
  int head_num = q.size(1);
  int q_len = q.size(2);
  int head_dim = q.size(3);
  int k_len = k.size(2);

  int head_stride = q.stride(1);

  auto out = torch::empty_like(q);

  float sm_scale = 1.0 / sqrt(head_dim) * M_LOG2E;

  // only for head_dim=64
  config::FlashConfig<cute::half_t> config;
  dim3 block = config.kThreadNum;
  dim3 grid((q_len + config.kBlockM - 1) / config.kBlockM, bs * head_num);
  int shm_size = config.kShmSize;
  auto partition_kernel = flash_forward<decltype(config)>;
  hipFuncSetAttribute(reinterpret_cast<const void*>(partition_kernel),
                       hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
  partition_kernel<<<grid, block, shm_size>>>(
      (void*)out.data_ptr(), (const void*)q.data_ptr(),
      (const void*)k.data_ptr(), (const void*)v.data_ptr(), head_stride, q_len,
      k_len, sm_scale);
  return out;
}