#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define A(i,j) matrix[(i)*n+(j)]
void random_init(int m,int n,float *matrix){
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            A(i,j)=2.0 *(float)drand48()-1.0;
        }
    }
}

void sgemm_cpu(float *A_ptr,float *B_ptr,float* C_ptr,
    const int m,const int k,const int n){
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            float tmp=0.f;
            for(int kk=0;kk<k;kk++){
                tmp+=A_ptr[k*i+kk]*B_ptr[n*kk+j];
            }
            C_ptr[i*n+j]=tmp;
        }
    }
}

template<unsigned int BLOCK_SIZE,unsigned int _K>
__global__ void cuda_sgemm(float* A_ptr,float* B_ptr, float* C_ptr,const int M,const int K,const int N){
    float* A_begin= A_ptr + blockIdx.y * blockDim.y * K;
    float* B_begin= B_ptr + blockIdx.x * blockDim.x;
    const int x=threadIdx.x+blockDim.x*blockIdx.x;
    const int y=threadIdx.y+blockDim.y*blockIdx.y;
    



    //两个shared mem 用来 sliding tile from A&B, 用C tile累加结果 
    // 1. 问题 是否有花括号初始化
    // 2.__device__修饰符在cuda当中存在吗 有意义吗
    __shared__ float a_shared=[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float b_shared=[BLOCK_SIZE][BLOCK_SIZE];

    //tile load & compute
    float sum=0.f;
    for(int s=0;s < K ;s+= blockDim.x){
        a_shared[threadIdx.y][threadIdx.x]=A_shared[s+threadIdx.x+threadIdx.y * K];
        a_shared[threadIdx.y][threadIdx.x]=B_shared[s*N+threadIdx.y*N+threadIdx.x ];//注意这里面不需要单独+s，因为是只复制过来竖条块
        __syncthreads();
        //这个地方注意转换
        for(int k=0;k<BLOCK_SIZE;k++){
            sum+=a_shared[threadIdx.y][k]*b_shared[k][threadIdx.x]; 
        }
        __syncthreads();
        
        
    }

    //滑动窗口完成
    C_ptr[y*N+x] =sum;
}

bool checkMatrix(float* a,float* b,const int m, const int n){
    return true;
}

int main(){
    //这个地方需要变小一点，对shared mem压力比较大
    constexpr int M=128;
    constexpr int N=128;
    constexpr int K=128;

    constexpr size_t mem_size_A=M*K*sizeof(float);
    constexpr size_t mem_size_B=K*N*sizeof(float);
    constexpr size_t mem_size_C=M*N*sizeof(float);

    float* maxtrix_A_host=(float*)malloc(mem_size_A);
    float* maxtrix_B_host=(float*)malloc(mem_size_B);
    float* matrix_C_host_calc=(float*)malloc(mem_size_C);
    float* matrix_C_device_calc=(float*)malloc(mem_size_C);


     
    float *matrix_C_d; 
    float *matrix_A_d; 
    float *matrix_B_d; 
    hipMalloc((void **)&matrix_C_d, mem_size_C);
    hipMalloc((void **)&matrix_A_d, mem_size_A);
    hipMalloc((void **)&matrix_B_d, mem_size_B);


    //init matrix
    {
        random_init(M,K,maxtrix_A_host);
        random_init(K,N,maxtrix_B_host);
        memset(matrix_C_host_calc,0,mem_size_C);
        memset(matrix_C_device_calc,0,mem_size_C);

        hipMemcpy(matrix_A_d,maxtrix_A_host,mem_size_A,hipMemcpyHostToDevice);
        hipMemcpy(matrix_B_d,maxtrix_B_host,mem_size_B,hipMemcpyHostToDevice);

    }

    constexpr int BLOCK=16;
    dim3 block(BLOCK,BLOCK);
    dim3 grid((M+BLOCK-1)/BLOCK,(N+BLOCK-1)/BLOCK);
    
    cuda_sgemm<<<grid,block>>>(matrix_A_d,matrix_B_d,matrix_C_d,M,K,N);


    {

    }

    return 0;
}




